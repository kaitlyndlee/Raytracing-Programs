#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

// Ray gen shader for ll00-rayGenOnly. No actual rays are harmed in the making
// of this shader. The pixel location is simply translated into a checkerboard
// pattern.

#include "deviceCode.h"
#include <optix_device.h>

// // OPTIX_RAYGEN_PROGRAM() is a simple macro defined in deviceAPI.h to add
// // standard code for defining a shader method. It puts:
// //   extern "C" __global__ void __raygen__##programName
// // in front of the program name given
// OPTIX_RAYGEN_PROGRAM(simpleRayGen)() {
//   // read in the program data set by the calling program hostCode.cpp using
//   // lloSbtRayGensBuild; see RayGenData in deviceCode.h
//   const RayGenData &self = owl::getProgramData<RayGenData>();
//   // Under the hood, OptiX maps rays generated in CUDA thread blocks to a pixel
//   // ID, where the ID is a 2D vector, 0 to frame buffer width-1, 0 to height-1
//   const vec2i pixelID = owl::getLaunchIndex();
//   if (pixelID == owl::vec2i(0)) {
//     // the first thread ID is always (0,0), so we can generate a message to show
//     // things are working
//     printf("%sHello OptiX From your First RayGen Program%s\n",
//            OWL_TERMINAL_CYAN, OWL_TERMINAL_DEFAULT);
//   }

//   // Generate a simple checkerboard pattern as a test. Note that the upper left
//   // corner is pixel (0,0).
//   int pattern = (pixelID.x / 8) ^ (pixelID.y / 8);
//   // alternate pattern, showing that pixel (0,0) is in the upper left corner
//   // pattern = (pixelID.x*pixelID.x + pixelID.y*pixelID.y) / 100000;
//   const vec3f color = (pattern & 1) ? self.color1 : self.color0;

//   // // find the frame buffer location (x + width*y) and put the "computed"
//   // result
//   // // there
//   // const int fbOfs = pixelID.x + self.fbSize.x * pixelID.y;
//   // self.fbPtr[fbOfs] = owl::make_rgba(color);

//   float color_vector[3] = {color.x, color.y, color.z};
//   self.pixmap[(pixelID.y * self.width + pixelID.x) * 3] = color_vector[0] * 255;
//   self.pixmap[(pixelID.y * self.width + pixelID.x) * 3 + 1] = color_vector[1] * 255;
//   self.pixmap[(pixelID.y * self.width + pixelID.x) * 3 + 2] = color_vector[2] * 255;
// }

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  vec3f &prd = owl::getPRD<vec3f>();

  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  
  // compute normal:
  const int   primID = optixGetPrimitiveIndex();
  const vec3i index  = self.index[primID];
  const vec3f &A     = self.vertex[index.x];
  const vec3f &B     = self.vertex[index.y];
  const vec3f &C     = self.vertex[index.z];
  const vec3f Ng     = normalize(cross(B-A,C-A));

  const vec3f rayDir = optixGetWorldRayDirection();
  prd = (.2f + .8f*fabs(dot(rayDir,Ng)))*self.color;
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  if (pixelID == owl::vec2i(0)) {
    printf("%sHello OptiX From your First RayGen Program%s\n",
           OWL_TERMINAL_CYAN,
           OWL_TERMINAL_DEFAULT);
  }
  PerRayData prd;
  owl::Ray ray;
  ray.origin = vec3f(0.f, 0.f, 0.f);
  int view_plane_center[3] = {0, 0, -1};

  ray.direction[0] = view_plane_center[0] - (self.camera_width / 2.0) + self.pixel_width * (pixelID.x + 0.5);
  ray.direction[1] = view_plane_center[1] + (self.camera_height / 2.0) - self.pixel_height * (pixelID.y + 0.5);
  ray.direction[2] = view_plane_center[2];

  ray.direction = normalize(ray.direction);

  vec3f color;
  owl::traceRay(/*accel to trace against*/self.world,
                /*the ray to trace*/ray,
                /*prd*/color);
    
  float color_vector[3] = {color.x, color.y, color.z};
  self.pixmap[(pixelID.y * self.width + pixelID.x) * 3] = color_vector[0] * 255;
  self.pixmap[(pixelID.y * self.width + pixelID.x) * 3 + 1] = color_vector[1] * 255;
  self.pixmap[(pixelID.y * self.width + pixelID.x) * 3 + 2] = color_vector[2] * 255;

}

OPTIX_MISS_PROGRAM(miss)() {
  const vec2i pixelID = owl::getLaunchIndex();

  const MissProgData &self = owl::getProgramData<MissProgData>();
  
  vec3f &prd = owl::getPRD<vec3f>();
  int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
  prd = vec3f(0, 0, 0);
}

OPTIX_INTERSECT_PROGRAM(Spheres)() {
      printf("%sIn Sphere intersect%s\n",
           OWL_TERMINAL_CYAN,
           OWL_TERMINAL_DEFAULT);
  const int primID = optixGetPrimitiveIndex();
  const auto &self = owl::getProgramData<SpheresList>().primitives[primID];

  const vec3f origin = optixGetWorldRayOrigin();
  const vec3f direction = optixGetWorldRayDirection();
  float hit_t = optixGetRayTmax();
  const float tmin = optixGetRayTmin();

  const vec3f origin_to_pos = origin - self.position;

  // float temp[3];

  // v3_subtract(temp, ray_o, sphere->position);

  // float a = ray_d[0] * ray_d[0] + ray_d[1] * ray_d[1] + ray_d[2]* ray_d[2];
   const float a = dot(direction, direction);
  // float b = 2 * v3_dot_product(ray_d, temp);
  const float b = 2 * dot(direction, origin_to_pos);
  // float c = v3_dot_product(temp, temp) - powf(sphere->radius, 2);
  const float c = dot(origin_to_pos, origin_to_pos) - (self.radius * self.radius);

  const float discriminant = b * b - 4 * a * c;

  if (discriminant < 0.f) {
    return;
  }
  else {
    // *distance = (-b - powf(discriminant, 0.5)) / (2.0 * a);

    // if (*distance < 0) {
    //   *distance = (-b + powf(discriminant, 0.5)) / (2.0 * a);
    // } 

    float temp = (-b + sqrtf(discriminant)) / (2.0 * a);
    if (temp < hit_t && temp > tmin) {
      hit_t = temp;
    }
    vec3f &prd = owl::getPRD<vec3f>();
    prd = self.diffuse_color;
  }
  if (hit_t < optixGetRayTmax()) {
    optixReportIntersection(hit_t, 0);
  }
}

OPTIX_BOUNDS_PROGRAM(Spheres)(const void *geomData,
                              box3f &primBounds,
                              const int primID) {
  const SpheresList &self = *(const SpheresList *) geomData;
  const Sphere sphere = self.primitives[primID];
  primBounds = box3f()
    .extend(sphere.position - sphere.radius)
    .extend(sphere.position + sphere.radius);

  printf("Position: [%f, %f, %f]\n", sphere.position.x, sphere.position.y, sphere.position.z);
  printf("radius: %f\n", sphere.radius);

}

OPTIX_CLOSEST_HIT_PROGRAM(Spheres)() {
  // const int primID = optixGetPrimitiveIndex();
  // const auto &self = owl::getProgramData<SpheresList>().primitives[primID];
  // PerRayData &prd = owl::getPRD<PerRayData>();
  // prd = self.diffuse_color;
}